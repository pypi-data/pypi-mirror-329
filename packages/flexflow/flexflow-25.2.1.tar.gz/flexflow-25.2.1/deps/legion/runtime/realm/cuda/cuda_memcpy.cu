#include "hip/hip_runtime.h"
/* Copyright 2024 Stanford University, NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <stdio.h>

#include <iostream>

#include "realm/cuda/cuda_memcpy.h"
#include "realm/point.h"

// The general formula for a linearized index is the following:
// I = \sum_{i=0}^{N} v_i (\prod_{j=0}^{i-1} D_j)
// This implies that the general form for a coordinate is:
// v_i = mod(div, D_j) where div is the floored dividend of all the dimensions D
// of the earlier dimensions, e.g. for 3D:
// I = z * D_y * D_x + y * D_x + x
// x = mod(I, D_x)                      = I % D_x
// y = mod(div(I, D_x), D_y)            = (I / D_x) % D_y
// z = mod(div(div((I,D_x),D_y), D_z)   = ((I / D_x) / D_y) % D_z

template <size_t N, typename Offset_t = size_t>
static __device__ inline void index_to_coords(Offset_t *coords, Offset_t index,
                                              const Offset_t *extents)
{
  size_t div = index;
#pragma unroll
  for(int i = 0; i < N - 1; i++) {
    size_t div_tmp = div / extents[i];
    coords[i] = div - div_tmp * extents[i];
    div = div_tmp;
  }
  coords[N - 1] = div;
}

template <size_t N, typename Offset_t = size_t>
static __device__ inline size_t coords_to_index(Offset_t *coords, const Offset_t *strides)
{
  size_t i = 0;
  size_t vol = 1;
  int d = 0;

#pragma unroll
  for(; d < N - 1; d++) {
    i += vol * coords[d];
    vol *= strides[d];
  }

  i += vol * coords[d];

  return i;
}

template <typename T, typename Offset_t = size_t>
static __device__ inline void memcpy_kernel_transpose(
    Realm::Cuda::MemcpyTransposeInfo<Offset_t> info, T* tile) {
  __restrict__ T *out_base = reinterpret_cast<T *>(info.dst);
  __restrict__ T *in_base = reinterpret_cast<T *>(info.src);
  const Offset_t tile_size = info.tile_size;
  const Offset_t tidx = threadIdx.x % tile_size;
  const Offset_t tidy = (threadIdx.x / tile_size) % tile_size;
  const Offset_t grid_dimx = ((info.extents[2] + tile_size - 1) / tile_size);
  const Offset_t grid_dimy = ((info.extents[1] + tile_size - 1) / tile_size);
  const Offset_t contig_bytes = info.extents[0];
  const Offset_t chunks = contig_bytes / sizeof(T);

  const Offset_t src_stride_x = info.src_strides[1] / contig_bytes;
  const Offset_t src_stride_y = info.src_strides[0] / contig_bytes;

  const Offset_t dst_stride_y = info.dst_strides[1] / contig_bytes;
  const Offset_t dst_stride_x = info.dst_strides[0] / contig_bytes;

  for(Offset_t block = blockIdx.x; block < grid_dimx * grid_dimy; block += gridDim.x) {
    Offset_t block_idx = block % grid_dimx;
    Offset_t block_idy = block / grid_dimx;

    Offset_t x_base = block_idx * tile_size * chunks + tidx;
    Offset_t y_base = block_idy * tile_size + tidy;

    __syncthreads();

    for(Offset_t block_offset = 0; block_offset < chunks * tile_size;
        block_offset += tile_size) {
      if(x_base + block_offset < info.extents[2] * chunks && y_base < info.extents[1]) {
        Offset_t in_tile_idx = tidx + (tile_size + 1) * tidy * chunks;

        // The purpose of this calculation is to handle XYZ -> ZYX case
        // where contig_bytes > sizeof(T)
        Offset_t x_base_idx =
            ((x_base / chunks) * (src_stride_x * chunks) + x_base % chunks);
        tile[in_tile_idx + block_offset] =
            in_base[x_base_idx + y_base * src_stride_y * chunks +
                    block_offset * src_stride_x];
      }
    }

    __syncthreads();

    x_base = block_idy * tile_size * chunks + tidx;
    y_base = block_idx * tile_size + tidy;

    for(Offset_t block_offset = 0; block_offset < chunks * tile_size;
        block_offset += tile_size) {
      if(x_base + block_offset < info.extents[1] * chunks && y_base < info.extents[2]) {
        Offset_t out_tile_idx =
            (tidy + (tile_size + 1) * ((tidx + block_offset) / chunks)) * chunks +
            (tidx + block_offset) % chunks;

        Offset_t x_base_idx =
            ((x_base / chunks) * (dst_stride_x * chunks) + x_base % chunks);

        out_base[x_base_idx + dst_stride_y * y_base * chunks +
                 block_offset * dst_stride_x] = tile[out_tile_idx];
      }
    }
  }
}

#define MAX_UNROLL (1)

template <typename T, size_t N, typename Offset_t = size_t>
static __device__ inline void
memcpy_affine_batch(Realm::Cuda::AffineCopyPair<N, Offset_t> *info,
                    size_t nrects, size_t start_offset = 0)
{
  Offset_t offset = blockIdx.x * blockDim.x + threadIdx.x - start_offset;
  const unsigned grid_stride = gridDim.x * blockDim.x;

  for(size_t rect = 0; rect < nrects; rect++) {
    Realm::Cuda::AffineCopyPair<N, Offset_t> &current_info = info[rect];
    const Offset_t vol = current_info.volume;
    __restrict__ T *dst = reinterpret_cast<T *>(current_info.dst.addr);
    __restrict__ T *src = reinterpret_cast<T *>(current_info.src.addr);

    while(offset < vol) {
      T tmp[MAX_UNROLL];
      unsigned i;

#pragma unroll
      for(i = 0; i < MAX_UNROLL; i++) {
        Offset_t src_coords[N];
        if((offset + i * grid_stride) >= vol) {
          break;
        }
        index_to_coords<N, Offset_t>(src_coords, offset + i * grid_stride,
                                     current_info.extents);
        const size_t src_idx =
            coords_to_index<N, Offset_t>(src_coords, current_info.src.strides);
        tmp[i] = src[src_idx];
      }
      for(unsigned j = 0; j < i; j++) {
        Offset_t dst_coords[N];

        index_to_coords<N, Offset_t>(dst_coords,
                                     (offset + j * grid_stride),
                                     current_info.extents);

        const size_t dst_idx =
            coords_to_index<N, Offset_t>(dst_coords, current_info.dst.strides);
        dst[dst_idx] = tmp[j];
      }

      offset += i * grid_stride;
    }

    // Skip this rectangle as it's covered by another thread
    // This can split the warp, and it may not coalesce again unless we sync them
    offset -= vol;
  }
}

/*
 * Scatter/gather points using indirection from/to dense buffer.
 * General assumptions:
 * 1. The src_ind/dst_ind buffer is dense and always has the same size
 * as the src/dst buffer (depending whether we are doing scatter or gather).
 * 2. src_ind_/dst_ind are accessed in a linear fashion with the base
 * type Point<N, Offset_t> per indirection element.
 * 3. src_ind/dst_ind do not have to be sorted but it should be
 * considered for coalesced access.
 *
 * TODO(apryakhin@): Consider handling ranges where src_ind/dst_ind
 * contain Rect<N, Offset_t> instead of Point<N Offset_t>.
 *
 * */

template <int N, typename T, typename DT, typename Offset_t = size_t>
static __device__ inline void
memcpy_indirect_points(Realm::Cuda::MemcpyIndirectInfo<3, Offset_t> info)
{
  Offset_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  __restrict__ T *dst_ind_base = reinterpret_cast<T *>(info.dst_ind_addr);
  __restrict__ T *src_ind_base = reinterpret_cast<T *>(info.src_ind_addr);

  Offset_t chunks = info.field_size / sizeof(DT);

  for(; offset < info.volume; offset += blockDim.x * gridDim.x) {
    Offset_t src_index = offset;
    if(info.src_ind_addr != 0) {
      Offset_t index = 0;
#pragma unroll
      for(int i = 0; i < N; i++) {
        index += src_ind_base[offset * N + i] * info.src_strides[i];
      }
      src_index = index;
    }

    Offset_t dst_index = offset;
    if(info.dst_ind_addr != 0) {
      Offset_t index = 0;
#pragma unroll
      for(int i = 0; i < N; i++) {
        index += dst_ind_base[offset * N + i] * info.dst_strides[i];
      }

      dst_index = index;
    }

    __restrict__ DT *dst =
        reinterpret_cast<DT *>(info.dst_addr + dst_index * info.field_size);
    __restrict__ DT *src =
        reinterpret_cast<DT *>(info.src_addr + src_index * info.field_size);
    for(Offset_t chunk_idx = 0; chunk_idx < chunks; chunk_idx++) {
      dst[chunk_idx] = src[chunk_idx];
    }
  }
}

template <int N, typename T, typename Offset_t = size_t>
static __device__ inline void
memfill_affine_batch(const Realm::Cuda::AffineFillInfo<N, Offset_t>& info)
{
  Offset_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned grid_stride = gridDim.x * blockDim.x;
  T fill_value = *reinterpret_cast<const T *>(info.fill_value);
  for(size_t rect = 0; rect < info.num_rects; rect++) {
    const Realm::Cuda::AffineFillRect<N, Offset_t> &current_info = info.subrects[rect];
    const Offset_t vol = current_info.volume;
    __restrict__ T *addr = reinterpret_cast<T *>(current_info.addr);
    while(offset < vol) {
      unsigned i = 0;
#pragma unroll
      for(i = 0; i < MAX_UNROLL; i++) {
        Offset_t coords[N];
        if((offset + i * grid_stride) >= vol) {
          break;
        }
        index_to_coords<N, Offset_t>(coords, offset + i * grid_stride,
                                     current_info.extents);
        const size_t idx = coords_to_index<N, Offset_t>(coords, current_info.strides);
        addr[idx] = fill_value;
      }
      offset += i * grid_stride;
    }
    // Skip this rectangle as it's covered by another thread
    // This can split the warp, and it may not coalesce again unless we sync them
    offset -= vol;
  }
}

#define MEMCPY_TEMPLATE_INST(type, dim, offt, name)                            \
  extern "C" __global__ __launch_bounds__(256, 4) void                         \
      memcpy_affine_batch##name(Realm::Cuda::AffineCopyInfo<dim, offt> info) { \
    memcpy_affine_batch<type, dim, offt>(info.subrects, info.num_rects);       \
  }

#define FILL_TEMPLATE_INST(type, dim, offt, name)                                        \
  extern "C" __global__ void fill_affine_batch##name(                                    \
      Realm::Cuda::AffineFillInfo<dim, offt> info) {                                     \
    memfill_affine_batch<dim, type, offt>(info);                                         \
  }

#define FILL_LARGE_TEMPLATE_INST(type, dim, offt, name)                                  \
  extern "C" __global__ void fill_affine_large##name(                                    \
      Realm::Cuda::AffineLargeFillInfo<dim, offt> info)                                  \
  {}

#define MEMCPY_TRANSPOSE_TEMPLATE_INST(type, offt, name)                                 \
  extern "C" __global__ __launch_bounds__(1024) void memcpy_transpose##name(             \
      Realm::Cuda::MemcpyTransposeInfo<offt> info)                                       \
  {                                                                                      \
    extern __shared__ type tile_shared_##name[];                                         \
    memcpy_kernel_transpose<type, offt>(info, tile_shared_##name);                       \
  }

#define MEMCPY_INDIRECT_TEMPLATE_INST(addr_type, data_type, dim, offt, name)             \
  extern "C" __global__ __launch_bounds__(256, 4) void memcpy_indirect##name(            \
      Realm::Cuda::MemcpyIndirectInfo<3, offt> info)                                     \
  {                                                                                      \
    memcpy_indirect_points<dim, addr_type, data_type, offt>(info);                       \
  }

#define INST_TEMPLATES(type, sz, dim, off)                                               \
  MEMCPY_TEMPLATE_INST(type, dim, off, dim##D_##sz)                                      \
  FILL_TEMPLATE_INST(type, dim, off, dim##D_##sz)                                        \
  FILL_LARGE_TEMPLATE_INST(type, dim, off, dim##D_##sz)                                  \
  MEMCPY_INDIRECT_TEMPLATE_INST(int, type, dim, off, dim##D_##sz##32)                    \
  MEMCPY_INDIRECT_TEMPLATE_INST(long long, type, dim, off, dim##D_##sz##64)

#define INST_TEMPLATES_FOR_TYPES(dim, off)                                     \
  INST_TEMPLATES(unsigned char, 8, dim, off)                                   \
  INST_TEMPLATES(unsigned short, 16, dim, off)                                 \
  INST_TEMPLATES(unsigned int, 32, dim, off)                                   \
  INST_TEMPLATES(unsigned long long, 64, dim, off)                             \
  INST_TEMPLATES(uint4, 128, dim, off)

#define INST_TEMPLATES_FOR_DIMS()                                                        \
  INST_TEMPLATES_FOR_TYPES(1, size_t)                                                    \
  INST_TEMPLATES_FOR_TYPES(2, size_t)                                                    \
  INST_TEMPLATES_FOR_TYPES(3, size_t)

INST_TEMPLATES_FOR_DIMS()

MEMCPY_TRANSPOSE_TEMPLATE_INST(unsigned char, size_t, 8)
MEMCPY_TRANSPOSE_TEMPLATE_INST(unsigned short, size_t, 16)
MEMCPY_TRANSPOSE_TEMPLATE_INST(unsigned int, size_t, 32)
MEMCPY_TRANSPOSE_TEMPLATE_INST(unsigned long long, size_t, 64)
MEMCPY_TRANSPOSE_TEMPLATE_INST(uint4, size_t, 128)
