#include "hip/hip_runtime.h"
//nvcc -o libgpu_point.so gpu_point.cu -shared -lcublas -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

// get_time(), time_since()
#include <ksgpu/time_utils.hpp>


__global__
void  fillA(float *dra_bore, float *ddec_bore, int n,float *dA)
{
  //int ra_ord=4;
  //int dec_ord=3;
  //int t_ord=2;
  int myi_off=blockIdx.x*blockDim.x+threadIdx.x;
  int nthread=blockDim.x*gridDim.x;
  for (int myind=myi_off;myind<n;myind+=nthread) {
    float myra=dra_bore[myind];
    float mydec=ddec_bore[myind];
    float tmp=myra*myra;

    dA[myind]=1.0;
    float t=2.0*(myind-(n-1))/(n-1.0)+1.0;
    dA[myind+1*n]=t;
    dA[myind+2*n]=t*t;


    dA[myind+6*n]=myra;
    dA[myind+7*n]=tmp;
    dA[myind+8*n]=myra*tmp;
    dA[myind+9*n]=tmp*tmp;
    tmp=mydec*mydec;
    dA[myind+3*n]=mydec;
    dA[myind+4*n]=tmp;
    dA[myind+5*n]=tmp*mydec;
    dA[myind+10*n]=myra*t;
    dA[myind+11*n]=mydec*t;
    dA[myind+12*n]=mydec*myra;
  }
}
/*--------------------------------------------------------------------------------*/
void print_arr(float *dvec, int n)
{
  float *hvec=(float *)malloc(sizeof(float)*n);
  if (hipMemcpy(hvec,dvec,n*sizeof(float),hipMemcpyDeviceToHost)!=hipSuccess) {
    fprintf(stderr,"Error copying from device in print_arr.\n");
    return;
  }
  for (int i=0;i<n;i++)
    printf("%5d %14.6g\n",i,hvec[i]);
  free(hvec);
      

}
/*--------------------------------------------------------------------------------*/
extern "C" {
void fillA_host(long *dra_bore,long *ddec_bore,int n, long *dA)
{
  float *raptr=(float *)(dra_bore[0]);
  float *decptr=(float *)(ddec_bore[0]);
  float *dAptr=(float *)(dA[0]);
  
  fillA<<<128,128>>>(raptr,decptr,n,dAptr);
  return;
  //printf("dec boresight:\n");
  //print_arr(ddec_bore,5);
  //fillA<<<128,128>>>(dra_bore,ddec_bore,n,dA);
}
}
/*--------------------------------------------------------------------------------*/
extern "C" {
void eval_fit(float *out,float *fitp, int n, int ndet, float *ra_bore, float *dec_bore)
{
  float *dA;
  int npar=13; //because we're super fragile right now
  if (hipMalloc((void **)&dA,sizeof(float)*n*npar)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  float *dra_bore;
  if (hipMalloc((void **)&dra_bore,sizeof(float)*n*npar)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");

  float *ddec_bore;
  if (hipMalloc((void **)&ddec_bore,sizeof(float)*n*npar)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");

  float *dfitp;
  if (hipMalloc((void **)&dfitp,sizeof(float)*ndet*npar)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");  
  if (hipMemcpy(dfitp,fitp,ndet*npar*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (hipMemcpy(dra_bore,ra_bore,n*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying ra bore data to device.\n");
  if (hipMemcpy(ddec_bore,dec_bore,n*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying dec bore data to device.\n");


  float *dout;
  if (hipMalloc((void **)&dout,sizeof(float)*ndet*n)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");  

  hipblasHandle_t handle;
  hipblasStatus_t stat;
  stat=hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
    return;
  }
  float one=1.0;
  float zero=0.0;

  hipDeviceSynchronize();
  for (int i=0;i<10;i++) {
      struct timeval tv = ksgpu::get_time();
    fillA<<<128,128>>>(dra_bore,ddec_bore,n,dA);
    stat=hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,n,ndet,npar,&one,dA,n,dfitp,ndet,&zero,dout,n);
    if (stat!=HIPBLAS_STATUS_SUCCESS) 
      printf("Error in sgemm.\n");
    hipDeviceSynchronize();
    double dt = ksgpu::time_since(tv);
    printf("Pointing reconstruction took %12.4g\n",dt);
  }
  if (hipMemcpy(out,dout,n*ndet*sizeof(float),hipMemcpyDeviceToHost)!=hipSuccess)
    fprintf(stderr,"Error copying out back to host.\n");
  hipFree(dA);
  hipFree(dra_bore);
  hipFree(ddec_bore);
  hipFree(dfitp);
}
}
